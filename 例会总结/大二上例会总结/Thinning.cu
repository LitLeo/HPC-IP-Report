#include "hip/hip_runtime.h"
// Thinning.cu
// 实现二值图像的细化算法。

#include"ErrorCode.h"
#include"Thinning.h"
#include<iostream>
#include<stdio.h>
using namespace std;

// 宏：DEF_BLOCK_X 和 DEF_BLOCK_Y
// 定义了默认的线程块尺寸。
#define DEF_BLOCK_X  32
#define DEF_BLOCK_Y   8


unsigned char lutthin1[] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 1, 1, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 1, 1, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 1, 1, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 1, 1, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 1, 1, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 1, 1, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 1, 1, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0};
unsigned char lutthin2[] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 0, 0, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 0, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 0, 1, 0, 1, 0, 1, 1, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 0, 1, 0, 1, 0, 1, 1, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 0, 1, 0, 1, 0, 1, 1, 0, 0, 1, 0, 0, 0};
unsigned char lutthin3[] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
unsigned char lutthin4[] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 1, 1, 1, 0, 0, 1, 1, 0, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 1, 1, 1, 0, 0, 1, 1, 0, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 1, 1, 1, 0, 0, 1, 1, 0, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 1, 1, 1, 0, 0, 1, 1, 0, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 1, 1, 1, 0, 0, 1, 1, 0, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 1, 1, 1, 0, 0, 1, 1, 0, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 1, 1, 1, 0, 0, 1, 1, 0, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 1, 1, 1, 0, 0, 1, 1, 0, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0};




// static 变量：_defLookTableData[18]
// 删除规则是由五十个 3×3 的模版组成，命名为细化查询表由于模版中的坐标
// 已经固定，所以在函数外直接定义这五十个模版的坐标，方便直接赋值。
static int _defLookTableData[18] = 
        {   
            -1,-1,  0,-1,  1,-1,  
            -1, 0,  0, 0,  1, 0,
            -1, 1,  0, 1,  1, 1,
        };

// static 变量：_defLookTableAttachedData[50][9]
// 按河边所给的 Thinning 算法文档中的描述，删除模版中有对应的数据，
// 在此利用 Template 中的 AttachedData （坐标点附带的数据）来存储
// 所给数据，且由于模版中的数据已经在算法文档中给出，所以在函数外
// 直接定义五十个个模版中的数据，方便为默认模版的附带数据数组赋值。
// 另外，因为图像是二值图像所以用 1 表示 HEIGHT PIXEL ,用 0 表示 
// LOW PIXEL。
static float _defLookTableAttachedData[50][9] = {
        // 模版内有 9 个数据
        // [0]
        {
            0.000000, 1.000000, 1.000000, 
            0.000000, 1.000000, 0.000000,
            0.000000, 0.000000, 0.000000
        },
        // [1]
        {
            0.000000, 0.000000, 1.000000,
            0.000000, 1.000000, 1.000000, 
            0.000000, 0.000000, 0.000000
        },
        // [2]
        {
            0.000000, 0.000000, 0.000000,
            0.000000, 1.000000, 1.000000,
            0.000000, 0.000000, 1.000000
        },
        // [3]
        {
            0.000000, 0.000000, 0.000000, 
            0.000000, 1.000000, 0.000000, 
            0.000000, 1.000000, 1.000000
        },
        //[4]
        {
            0.000000, 0.000000, 0.000000,
            0.000000, 1.000000, 0.000000,
            1.000000, 1.000000, 0.000000
        },
        // [5]
        {
            0.000000, 0.000000, 0.000000,
            1.000000, 1.000000, 0.000000,
            1.000000, 0.000000, 0.000000
        },
        // [6]
        {
            1.000000, 0.000000, 0.000000,
            1.000000, 1.000000, 0.000000, 
            0.000000, 0.000000, 0.000000
        },
        // [7]
        {
            1.000000, 1.000000, 0.000000,
            0.000000, 1.000000, 0.000000,
            0.000000, 0.000000, 0.000000
        },
        // 模版内有四个数据
        // [8]
        {
            1.000000, 1.000000, 1.000000, 
            0.000000, 1.000000, 0.000000,
            0.000000, 0.000000, 0.000000
        },
        // [9]
        {
            0.000000, 1.000000, 1.000000, 
            0.000000, 1.000000, 1.000000, 
            0.000000, 0.000000, 0.000000
        },
        // [10]
        {
            0.000000, 0.000000, 1.000000,
            0.000000, 1.000000, 1.000000,
            0.000000, 0.000000, 1.000000
        },
        // [11]
        {
            0.000000, 0.000000, 0.000000,
            0.000000, 1.000000, 1.000000,
            0.000000, 1.000000, 1.000000
        },
        // [12]
        {
            0.000000, 0.000000, 0.000000,
            0.000000, 1.000000, 0.000000, 
            1.000000, 1.000000, 1.000000
        },
        // [13]
        {
            0.000000, 0.000000, 0.000000,
            1.000000, 1.000000, 0.000000,
            1.000000, 1.000000, 0.000000
        },
        // [14]
        {
            1.000000, 0.000000, 0.000000,
            1.000000, 1.000000, 0.000000,
            1.000000, 0.000000, 0.000000
        },
        // [15]
        {
            1.000000, 1.000000, 0.000000,
            1.000000, 1.000000, 0.000000,
            0.000000, 0.000000, 0.000000
        },
        // [16]
        {
            0.000000, 1.000000, 0.000000,
            1.000000, 1.000000, 0.000000, 
            1.000000, 0.000000, 0.000000
        },
        // [17]
        {
            0.000000, 1.000000, 0.000000,
            0.000000, 1.000000, 1.000000,
            0.000000, 0.000000, 1.000000
        },
        // [18]
        {
            1.000000, 1.000000, 0.000000,
            0.000000, 1.000000, 1.000000,
            0.000000, 0.000000, 0.000000
        },
        // [19]
        {
            0.000000, 1.000000, 1.000000,
            1.000000, 1.000000, 0.000000,
            0.000000, 0.000000, 0.000000
        },
        // 模版内有5个数据
        // [20]
        {
            1.000000, 1.000000, 1.000000,
            0.000000, 1.000000, 1.000000,
            0.000000, 0.000000, 0.000000
        },
        // [21]
        {
            0.000000, 1.000000, 1.000000,
            0.000000, 1.000000, 1.000000,
            0.000000, 0.000000, 1.000000
        },
        // [22]
        {
            0.000000, 0.000000, 1.000000,
            0.000000, 1.000000, 1.000000,
            0.000000, 1.000000, 1.000000
        },
        // [23]
        {
            0.000000, 0.000000, 0.000000,
            0.000000, 1.000000, 1.000000,
            1.000000, 1.000000, 1.000000
        },
        // [24]
        {
            0.000000, 0.000000, 0.000000,
            1.000000, 1.000000, 0.000000,
            1.000000, 1.000000, 1.000000
        },
        // [25]
        {
            1.000000, 0.000000, 0.000000,
            1.000000, 1.000000, 0.000000,
            1.000000, 1.000000, 0.000000
        },
        // [26]
        {
            1.000000, 1.000000, 0.000000, 
            1.000000, 1.000000, 0.000000,
            1.000000, 0.000000, 0.000000
        },
        // [27]
        {
            1.000000, 1.000000, 1.000000,
            1.000000, 1.000000, 0.000000,
            0.000000, 0.000000, 0.000000
        },
        // [28]
        {
            1.000000, 1.000000, 0.000000,
            0.000000 ,1.000000, 1.000000, 
            0.000000, 0.000000, 1.000000
        },
        // [29]
        {
            0.000000, 1.000000, 1.000000, 
            1.000000, 1.000000, 0.000000,
            1.000000, 0.000000, 0.000000
        },
        // 模板内有六个数据
        // [30]
        {
            1.000000, 1.000000, 1.000000,
            0.000000, 1.000000, 1.000000,
            0.000000, 0.000000, 1.000000
        },
        // [31]
        {
            0.000000, 1.000000, 1.000000, 
            0.000000, 1.000000, 1.000000,
            0.000000, 1.000000, 1.000000
        },
        // [32]
        {
            0.000000, 0.000000, 1.000000,
            0.000000, 1.000000, 1.000000,
            1.000000, 1.000000, 1.000000
        },
        // [33]
        {
            0.000000, 0.000000, 0.000000,
            1.000000, 1.000000, 1.000000,
            1.000000, 1.000000, 1.000000
        },
        // [34]
        {
            1.000000, 0.000000, 0.000000,
            1.000000, 1.000000, 0.000000,
            1.000000, 1.000000, 1.000000
        },
        // [35]
        {
            1.000000, 1.000000, 0.000000, 
            1.000000, 1.000000, 0.000000,
            1.000000, 1.000000, 0.000000
        },
        // [36]
        {
            1.000000, 1.000000, 1.000000,
            1.000000, 1.000000, 0.000000,
            1.000000, 0.000000, 0.000000
        },
        // [37]
        {
            1.000000, 1.000000, 1.000000,
            1.000000, 1.000000, 1.000000,
            0.000000, 0.000000, 0.000000
        },
        // 模版内有7个像素
        // [38]
         {
            1.000000, 1.000000, 1.000000,
            0.000000, 1.000000, 1.000000,
            0.000000, 0.000000, 1.000000
        },
        // [39]
        {
            0.000000, 1.000000, 1.000000, 
            0.000000, 1.000000, 1.000000,
            1.000000, 1.000000, 1.000000
        },
        // [40]
        {
            0.000000, 0.000000, 1.000000,
            1.000000, 1.000000, 1.000000,
            1.000000, 1.000000, 1.000000
        },
        // [41]
        {
            1.000000, 0.000000, 0.000000,
            1.000000, 1.000000, 1.000000,
            1.000000, 1.000000, 1.000000
        },
        // [42]
        {
            1.000000, 1.000000, 0.000000,
            1.000000, 1.000000, 0.000000,
            1.000000, 1.000000, 1.000000
        },
        // [43]
        {
            1.000000, 1.000000, 1.000000, 
            1.000000, 1.000000, 0.000000,
            1.000000, 1.000000, 0.000000
        },
        // [44]
        {
            1.000000, 1.000000, 1.000000,
            1.000000, 1.000000, 1.000000,
            1.000000, 0.000000, 0.000000
        },
        // [45]
        {
            1.000000, 1.000000, 1.000000,
            1.000000, 1.000000, 1.000000,
            0.000000, 0.000000, 1.000000
        },
        // 模版内有八个元素
        // [46]
        {
            1.000000, 1.000000, 1.000000,
            1.000000, 1.000000, 1.000000,
            1.000000, 0.000000, 1.000000
        },
        // [47]
        {
            1.000000, 1.000000, 1.000000,
            0.000000, 1.000000, 1.000000,
            1.000000, 1.000000, 1.000000
        },
        // [48]
        {
            1.000000, 0.000000, 1.000000,
            1.000000, 1.000000, 1.000000,
            1.000000, 1.000000, 1.000000
        },
        // [49]
        {
            1.000000, 1.000000, 1.000000,
            1.000000, 1.000000, 0.000000,
            1.000000, 1.000000, 1.000000
        }
};


// static 变量：_defTemplateData[7][32]
// 由于模版中的坐标已经固定，所以在函数外直接定义这七个模版坐标,
// 方便为默认模版坐标赋值。
static int _defTemplateData[7][32] = {
        // 3*4（1） 模版一
        // [0]
        {   -1,-1,  0,-1,  1,-1,  
            -1, 0,  0, 0,  1, 0,
            -1, 1,  0, 1,  1, 1,
            -1, 2,  0, 2,  1, 2
        },
        // 3*4（2） 模版二
        // [1]
        {   -1,-2,  0,-2,  1,-2,
            -1,-1,  0,-1,  1,-1,
            -1, 0,  0, 0,  1, 0,
            -1, 1,  0, 1,  1, 1
        },
        // 3*4（3） 模版三
        // [2]
        {   -1,-1,  0,-1,  1,-1,  
            -1, 0,  0, 0,  1, 0,
            -1, 1,  0, 1,  1, 1,
            -1, 2,  0, 2,  1, 2
        },
        // 4*3（1） 模版四
        // [3]
        {  -1,-1,  0,-1,  1,-1,  2,-1,
           -1, 0,  0, 0,  1, 0,  2, 0,
           -1, 1,  0, 1,  1, 1,  2, 1
        },
        // 4*3（2） 模版五
        // [4]
        {  -1,-1,  0,-1,  1,-1,  2,-1,
           -1, 0,  0, 0,  1, 0,  2, 0,
           -1, 1,  0, 1,  1, 1,  2, 1
        },
        // 4*3（3） 模版六
        // [5]
        {  -2,-1,  -1,-1,  0,-1,  1,-1,
           -2, 0,  -1, 0,  0, 0,  1, 0,
           -2, 1,  -1, 1,  0, 1,  1, 1
        },
        // 4*4（1） 模版七
        // [6]
        {  -1,-1,  0,-1,  1,-1,  2,-1,
           -1, 0,  0, 0,  1, 0,  2, 0,
           -1, 1,  0, 1,  1, 1,  2, 1,
           -1, 2,  0, 2,  1, 2,  2, 2
        }
};

// static 变量：_defTemplateAttachedData[7][50]
// 按河边所给的 Thinning 算法文档中的描述，删除模版中有对应的数据，
// 在此利用 Template 中的 AttachedData （坐标点附带的数据）来存储
// 所给数据，且由于模版中的数据已经在算法文档中给出，所以在函数外
// 直接定义七个模版中的数据，方便为默认模版的附带数据数组赋值。
// 另外，因为图像是二值图像并且所给数据中有无关数据（即忽略该点数
// 据对模版的影响）,所以用 1 表示HEIGHT PIXEL ,用 0 表示 LOW PIXEL，
// 用 -1 表示无关数据。
static float _defTemplateAttachedData[7][16] = {
        // 模版一数据
       {  -1.000000,  0.000000, -1.000000,
           1.000000,  1.000000,  1.000000,
           1.000000,  1.000000,  1.000000,
          -1.000000,  0.000000, -1.000000
       },
        // 模版二数据
       {  -1.000000,  0.000000,  0.000000,
           1.000000,  1.000000,  0.000000,
           0.000000,  1.000000,  0.000000,
           0.000000,  0.000000, -1.000000
       },
        // 模版三数据
       {  -1.000000,  0.000000,  0.000000,
           0.000000,  1.000000,  0.000000,
           0.000000,  1.000000,  1.000000,
           0.000000,  0.000000, -1.000000
       },
        // 模版四数据
       {  -1.000000,  0.000000,  0.000000,  0.000000,
           0.000000,  1.000000,  1.000000,  0.000000,
           0.000000,  0.000000,  1.000000, -1.000000
       },
        // 模版五数据
       {  -1.000000,  1.000000,  1.000000, -1.000000,
           0.000000,  1.000000,  1.000000,  0.000000,
          -1.000000,  1.000000,  1.000000, -1.000000
       },
        // 模版六数据
       {   0.000000,  0.000000,  0.000000, -1.000000,
           0.000000,  1.000000,  1.000000,  0.000000,
          -1.000000,  1.000000,  0.000000,  0.000000
       },
        // 模版七数据
       {   0.000000,  0.000000,  0.000000,  0.000000,
           0.000000,  1.000000,  1.000000,  0.000000,
           0.000000,  1.000000,  1.000000,  0.000000,
           0.000000,  0.000000,  0.000000,  0.000000
       }
};

// Kernel 函数：_thinningKer（实现Thining算法操作）
// 在调用此 kernel 函数时，已经将输入图像拷贝到输出图像
// 所以在参数里直接对输出图像进行操作
static __global__ void                // Kernel 函数无返回值
_thinningKer(
        ImageCuda outimg,             // 输出图像
        TemplateCuda **tableTplCuda,  // 细化查询表
        TemplateCuda **tplCuda,       // 删除模板数组
        int deleteTemplateLen ,       // 删除模版数组的长度
        int lookTableLen,             // 细化查询表的长度
        int *TabArray                 // 标记数组
);

// Kernel 函数：_deleteImageKer（实现删除算法操作）
// 调用此函数，根据标记数组 TabArray 的值判断该点是否应该被删除,
// 因为会迭代调用此核函数，isIteration 为结束的迭代的标记。
static __global__ void
_deleteImageKer(
        ImageCuda outimg,             // 输出图像
        int *TabArray,                // 标记数组
        bool *isIteration             // 结束迭代的标记

);


static __global__ void _thin_MATLABLike_FsubKer(
        ImageCuda outimg,
        ImageCuda tempimg,
        char *lutthin1_dev,
        char *lutthin2_dev,
        char *lutthin3_dev
        );


static __global__ void _thin_MATLABLike_SsubKer(
        ImageCuda tempimg,
        ImageCuda outimg,
        char *lutthin1_dev,
        char *lutthin2_dev,
        char *lutthin4_dev,
        int *dev_changedCount
        );

// 构造函数：Thinning
__host__ Thinning::Thinning()
{
    setLookTableLen(50);              // 初始化删除模版数组
                                      // 的长度为默认的 50。
    setDeleteTemplateLen(7);          // 初始化细化查询表的长度为 7。
}

// 成员方法：getLookTableLen
__host__ int Thinning::getLookTableLen() const
{
    // 如果 lookTableLen 不为负值，则返回lookTableLen.
    // if (lookTableLen >= 0)
        return lookTableLen;
}


// 成员方法：setLookTableLen
__host__ void Thinning::setLookTableLen(int _lookTableLen)
{
    // if (_lookTableLen >= 0)
        lookTableLen = _lookTableLen;
}

 // 成员方法：getDeleteTemplateLen
 __host__ int Thinning::getDeleteTemplateLen() const
 {
    // 如果 deleteTemplateLen 不为负值，则返回deleteTemplateLen。
    // if (deleteTemplateLen >= 0)
        return deleteTemplateLen;
 }

// 成员方法：setDeleteTemplateLen
 __host__ void Thinning::setDeleteTemplateLen(int _deleteTemplateLen)
 {
    // if (_deleteTemplateLen >= 0)
        deleteTemplateLen = _deleteTemplateLen;
 }

// Kernel 函数：_thinningKer（实现细化算法操作）
static __global__ void _thinningKer(ImageCuda outimg, 
                                TemplateCuda **tableTplCuda, 
                                TemplateCuda **tplCuda, 
                                int deleteTemplateLen,
                                int lookTableLen,
                                int *TabArray)
{
    // dstc 和 dstr 分别表示线程处理的像素点的坐标的 x 和 y 分量 （其中，
    // c 表示 column， r 表示 row ）。
    int dstc = blockIdx.x * blockDim.x + threadIdx.x;
    int dstr = blockIdx.y * blockDim.y + threadIdx.y;

    // 检查第一个像素点是否越界，如果越界，则不进行处理，一方面节省计算
    // 资源，另一方面防止由于段错误导致程序崩溃。
    if (dstc >= outimg.imgMeta.width || dstr >= outimg.imgMeta.height)
        return;

    // 用来保存临时像素点的坐标的 x 和 y 分量。
    int dx, dy;     
    
    // 定义输出图像位置的指针。
    unsigned char *outptr;
    
    // 存放模版像素点所在位置的指针。
    unsigned char *pixel;
    
	// 定义标记变量，其值为 1 或 0 ，当目标像素点不符合某一模版时，其值
	// 置为 1，且跳出当前模版循环，继续遍历下一模版。直至目标点符合某一
    // 模版或遍历模版结束。
    int sign ;
    
    // 获取当前当前像素点在图像中的相对位置。
    int curpos = dstr * outimg.pitchBytes + dstc;

    // 获取对应的第一个输出图像的位置。
    outptr = outimg.imgMeta.imgData + curpos;

    // 初始化该像素点对应标记数组的值为 0 ，即不删除。
    TabArray[curpos] = 0;

    // 因为无论是细化查询表还是删除模版数组，目标点的值都为 1 ，
    // 所以像素为 0 的像素点不执行细化查询表和删除模版数组的遍历。
    if (*outptr > 0)
    {  
        // 扫描细化查询表内的所有模板。
        for (int i=0; i<lookTableLen; i++)
        {
            // 每一个模版循环开始时，定义 sign 的值为 0，若该点不符合该模版，则使 sign = 1，
            // 并跳出当前模版循环，跳出后对 sign 的值进行判断，如果 sign 的值为 0， 则代表
            // 该点符合该模版，使其对应的标记数组的值为 1。
            sign = 0;

            // 扫描细化查询表范围内的每个输入图像的像素点。
            for (int j = 0; j < tableTplCuda[i]->tplMeta.count; j++)
            {
                // 计算当前模板位置所在像素的 x 和 y 分量，模板使用相邻的两个下标的
                // 数组表示一个点，所以使当前模板位置的指针作加一操作 。
                dx = dstc + (tableTplCuda[i]->tplMeta.tplData[j*2]);
                dy = dstr + (tableTplCuda[i]->tplMeta.tplData[j*2+1]);
    
                // 先判断当前像素的 x 分量和 y 分量是否越界，如果越界，则跳过，扫描
                // 下一个模板点。
                if (dx >= 0 && dx < outimg.imgMeta.width && 
                    dy >= 0 && dy < outimg.imgMeta.height) 
                {
                    // 根据 dx 和 dy 获取第一个像素的位置。
                    pixel = outimg.imgMeta.imgData + dx + dy * outimg.imgMeta.width;
                        
                    // 将目标点模版范围的像素点的值转化成二值并与模版附属数据值进行
                    // 匹配。
                    if ( (*pixel > 0) != tableTplCuda[i]->attachedData[j] )
                    {
                        // 如果某一点不符合，则该模版不匹配该像素点， 将 sign 置为 1 
                        //并跳出该模版的循环，进入下一循环。
                         sign = 1; 
                         break;
                    }
                }  
            }

            // 遍历完一个模版后，如果 sign 值为 0，则代表目标点与该模版相匹配，将该点
            // 对应的标记数组的值置为 1，并跳出模版循环。
            if(sign == 0)
            {
              TabArray[curpos] = 1;
              break;
            }
        }

        // 当模版遍历结束时，如果 TabArray[curpos] 的值为 1，代表目标点与细化查询表
        // 内的所有模版都不匹配，则进行删除模版的循环，如果当前像素满足任何一个模版，
        // 则不删除此像素（即使其满足步骤 2 中的删除条件）。
        if (TabArray[curpos] == 1)
        {
            for (int i=0; i<deleteTemplateLen; i++)
            {
                // 重复使用 sign ，功能与前者相同。
                sign = 0;

                // 扫描模板范围内的每个输入图像的像素点。
                for (int j = 0; j < tplCuda[i]->tplMeta.count; j++)
                {
                    // 计算当前模板位置所在像素的 x 和 y 分量，模板使用相邻的两个下标的
                    // 数组表示一个点，所以使当前模板位置的指针作加一操作 。
                    dx = dstc + (tplCuda[i]->tplMeta.tplData[j*2]);
                    dy = dstr + (tplCuda[i]->tplMeta.tplData[j*2+1]);
            
                    // 先判断当前像素的 x 分量和 y 分量是否越界，如果越界，则跳过，扫描
                    // 下一个模板点。
                    if (dx >= 0 && dx < outimg.imgMeta.width && 
                    dy >= 0 && dy < outimg.imgMeta.height) 
                    {
                        // 根据 dx 和 dy 获取第一个像素的位置。
                        pixel = outimg.imgMeta.imgData + dx + dy * outimg.imgMeta.width;
                
                        // 将目标点模版范围的像素点的值转化成二值并与模版附属数据值
                        // 进行匹配。
                        if ( tplCuda[i]->attachedData[j]>-1 && 
                        (*pixel > 0) != tplCuda[i]->attachedData[j] )
                        {
                            // 如果某一点不符合，则该模版不匹配该像素点，将 sign 置为 1 
                            // 并跳出该模版的循环，进入下一循环。
                            sign = 1;
                            break;
                        }
                    }  
                }
                // 遍历完一个模版后，如果 sign 值为 0，则代表目标点与该模版相匹配，
                // 将 TabArray[curpos] 置为0，并跳出模版循环。
                if(sign == 0)
                {
                    TabArray[curpos] = 0;
                    break;
                }
            }
        } 
    } 
 }  

// Kernel 函数：_deleteImageKer（实现删除图像算法操作）
static __global__ void _deleteImageKer(ImageCuda outimg, int *TabArray, bool *isIteration)
 {
    // dstc 和 dstr 分别表示线程处理的像素点的坐标的 x 和 y 分量 （其中，
    // c 表示 column， r 表示 row）。
    int dstc = blockIdx.x * blockDim.x + threadIdx.x;
    int dstr = blockIdx.y * blockDim.y + threadIdx.y;

    // 检查第一个像素点是否越界，如果越界，则不进行处理，一方面节省计算
    // 资源，另一方面防止由于段错误导致程序崩溃。
    if (dstc >= outimg.imgMeta.width || dstr >= outimg.imgMeta.height)
        return;

    // 定义输出图像位置的指针。
    unsigned char *outptr;

    // 获取当前像素点在图像中的相对位置。
    int curpos = dstr * outimg.pitchBytes + dstc;

    // 获取当前像素点在图像中的绝对位置。
    outptr = outimg.imgMeta.imgData + curpos ;

    // 如果该点对应的标记数组的值为 1，则删除干像素点并将 TabArray[curpos]
    // 值重新置为 0。
    if (TabArray[curpos] == 1)
    {

        *outptr = 0;
        // 标记迭代的变量，已初始化置为 false，若图像中有至少一个一个像素点
        // 被删除，则置 isIteration 为 true ，表示继续迭代。
        *isIteration = true;
        TabArray[curpos] = 0;
    }
 }


static __global__ void _thin_MATLABLike_FsubKer(
        ImageCuda outimg,
        ImageCuda tempimg,
        char *lutthin1_dev,
        char *lutthin2_dev,
        char *lutthin3_dev
        )
{

// printf("1 ");
    // dstc 和 dstr 分别表示线程处理的像素点的坐标的 x 和 y 分量 （其中，
    // c 表示 column， r 表示 row）。
    int dstc = blockIdx.x * blockDim.x + threadIdx.x;
    int dstr = blockIdx.y * blockDim.y + threadIdx.y;

    // 检查第一个像素点是否越界，如果越界，则不进行处理，一方面节省计算
    // 资源，另一方面防止由于段错误导致程序崩溃。
    if (dstc >= outimg.imgMeta.width-1 || dstr >= outimg.imgMeta.height-1 ||
        dstc < 1 || dstr < 1)
        return;
        // printf("2 ");
        

    // 定义输出图像位置的指针。
    unsigned char *outptr;

    // 获取当前像素点在图像中的相对位置。
    // int curpos = dstr * outimg.imgMeta.width + dstc;
    int curpos = dstr *  outimg.pitchBytes + dstc;

    // 获取当前像素点在图像中的绝对位置。
    outptr = outimg.imgMeta.imgData + curpos ;
//printf("1 ");
if(*outptr > 0)
{
    int index = 0;
//printf("%d ", *outptr);
    if(outimg.imgMeta.imgData[dstc-1 + (dstr-1) * outimg.pitchBytes] > 0) index += 1;
    if(outimg.imgMeta.imgData[dstc-1 + (dstr  ) * outimg.pitchBytes] > 0) index += 2;
    if(outimg.imgMeta.imgData[dstc-1 + (dstr+1) * outimg.pitchBytes] > 0) index += 4;
// printf("1 ");
    if(outimg.imgMeta.imgData[dstc + (dstr-1) * outimg.pitchBytes] > 0) index += 8;
    if(outimg.imgMeta.imgData[dstc + (dstr) * outimg.pitchBytes] > 0) index += 16;
    if(outimg.imgMeta.imgData[dstc + (dstr+1) * outimg.pitchBytes] > 0) index += 32;

    if(outimg.imgMeta.imgData[dstc+1 + (dstr-1) * outimg.pitchBytes] > 0) index += 64;
    if(outimg.imgMeta.imgData[dstc+1 + (dstr) * outimg.pitchBytes] > 0) index += 128;
    if(outimg.imgMeta.imgData[dstc+1 + (dstr+1) * outimg.pitchBytes] > 0) index += 256;

    unsigned char replacedPix1 = lutthin1_dev[index];
    unsigned char replacedPix2 = lutthin2_dev[index];
    unsigned char replacedPix3 = lutthin3_dev[index];
    //printf("2 ");
    tempimg.imgMeta.imgData[curpos] = *outptr && !(replacedPix1 && replacedPix2 && replacedPix3);
    }
    else
    {
    tempimg.imgMeta.imgData[curpos] = 0;
    }
}

static __global__ void _thin_MATLABLike_SsubKer(
        ImageCuda tempimg,
        ImageCuda outimg,
        char *lutthin1_dev,
        char *lutthin2_dev,
        char *lutthin4_dev,
        int *dev_changedCount
        )
{
// printf("%d ", *dev_changedCount);
    // *dev_changedCount = 0;
    // dstc 和 dstr 分别表示线程处理的像素点的坐标的 x 和 y 分量 （其中，
    // c 表示 column， r 表示 row）。
    int dstc = blockIdx.x * blockDim.x + threadIdx.x;
    int dstr = blockIdx.y * blockDim.y + threadIdx.y;

    // 检查第一个像素点是否越界，如果越界，则不进行处理，一方面节省计算
    // 资源，另一方面防止由于段错误导致程序崩溃。
    if (dstc >= tempimg.imgMeta.width-1 || dstr >= tempimg.imgMeta.height-1 ||
        dstc < 1 || dstr < 1)
        return;

    // 定义输出图像位置的指针。
    unsigned char *outptr;

    // 获取当前像素点在图像中的相对位置。
    // int curpos = dstr * outimg.imgMeta.width + dstc;
    int curpos = dstr * outimg.pitchBytes + dstc;
    
    // 获取当前像素点在图像中的绝对位置。
    outptr = tempimg.imgMeta.imgData + curpos ;

    int index = 0;

    if(tempimg.imgMeta.imgData[dstc-1 + (dstr-1) * tempimg.pitchBytes] > 0) index += 1;
    if(tempimg.imgMeta.imgData[dstc-1 + (dstr  ) * tempimg.pitchBytes] > 0) index += 2;
    if(tempimg.imgMeta.imgData[dstc-1 + (dstr+1) * tempimg.pitchBytes] > 0) index += 4;

    if(tempimg.imgMeta.imgData[dstc + (dstr-1) * tempimg.pitchBytes] > 0) index += 8;
    if(tempimg.imgMeta.imgData[dstc + (dstr  ) * tempimg.pitchBytes] > 0) index += 16;
    if(tempimg.imgMeta.imgData[dstc + (dstr+1) * tempimg.pitchBytes] > 0) index += 32;

    if(tempimg.imgMeta.imgData[dstc+1 + (dstr-1) * tempimg.pitchBytes] > 0) index += 64;
    if(tempimg.imgMeta.imgData[dstc+1 + (dstr  ) * tempimg.pitchBytes] > 0) index += 128;
    if(tempimg.imgMeta.imgData[dstc+1 + (dstr+1) * tempimg.pitchBytes] > 0) index += 256;

    unsigned char replacedPix1 = lutthin1_dev[index];
    unsigned char replacedPix2 = lutthin2_dev[index];
    unsigned char replacedPix4 = lutthin4_dev[index];

    unsigned char niv = *outptr && !(replacedPix1 && replacedPix2 && replacedPix4);

    if (niv != (outimg.imgMeta.imgData[curpos]>0))
    {
        if (niv)
            outimg.imgMeta.imgData[curpos] = 255;
        else
            outimg.imgMeta.imgData[curpos] = niv;
        // (*dev_changedCount) ++;
        //  printf("%d ", *dev_changedCount);
        atomicAdd(dev_changedCount,1);
    }
  
}



 
 
// 成员方法：thinEdge
__host__ int Thinning::thinEdge(Image *inimg, Image *outimg)
{
    int errcode;  // 局部变量，错误码。
    dim3 gridsize;
    dim3 blocksize;
    
    // 由于将细化查询表和删除模版的数据拷进 device 比较复杂,
    // 在这里对细化查询表和模版数组各定义三个二维模版指针变量来执行此操作。
    TemplateCuda *tplCudaHost[7] = {NULL}; 
    TemplateCuda **tplCuda;
    TemplateCuda *tplCudaTemp[7];

    TemplateCuda *lookTableCudaHost[50] = {NULL};
    TemplateCuda **lookTableCuda;
    TemplateCuda *lookTableCudaTemp[50];

    int *TabArray;                    // 标记数组
    bool *isIteration = new bool();   // 位于 host 端的标记迭代的变量。
    bool *dev_isIteration;            // 位于 device 端的标记迭代的变量。

    // 检查输入图像，输出图像，以及模板是否为空。
    if (inimg == NULL || outimg == NULL || deleteTemplate == NULL)
        return NULL_POINTER;

    // 将输入输出图像拷贝到 Device 内存中。
    errcode = ImageBasicOp::copyToCurrentDevice(inimg, outimg);
    if (errcode != NO_ERROR)
        return errcode;          
        
     // 为细化查询表内的模版开空间并为其赋值。
    for (int i = 0; i < lookTableLen; ++i)
    {
        TemplateBasicOp::newTemplate(&lookTable[i]);
        TemplateBasicOp::makeAtHost(lookTable[i], 9);
        lookTableCudaHost[i] = TEMPLATE_CUDA(lookTable[i]);
        for (int j = 0; j < lookTable[0]->count; ++j)
        {
            // 为模版的坐标数据赋值。
            lookTable[i]->tplData[2*j] = _defLookTableData[2*j];
            lookTable[i]->tplData[2*j +1] = _defLookTableData[2*j + 1];
            
            // 为模版的附属数据赋值。
            lookTableCudaHost[i]->attachedData[j] = _defLookTableAttachedData[i][j];
        }
    }

    // 为删除模版数组内的模版赋值。因为前六个模版的大小相同，
    // 所以用一个 for 循环为其开空间。
    for (int i=0; i<deleteTemplateLen; i++)
    {
        TemplateBasicOp::newTemplate(&(deleteTemplate[i]));
        TemplateBasicOp::makeAtHost(deleteTemplate[i], 12);
    }
    
    // 第七个模版的大小为 16，再次单独为其开空间。
    TemplateBasicOp::newTemplate(&deleteTemplate[6]);
    TemplateBasicOp::makeAtHost(deleteTemplate[6], 16);
    
    // 通过调用前面定义的 _defTemplateData 数组和 _defTemplateAttachedData 
    // 数组为七个删除模版的坐标数据和附属数据赋值。
    for(int i=0; i<deleteTemplateLen; i++)
    {
        tplCudaHost[i] = TEMPLATE_CUDA(deleteTemplate[i]);
        for(int j=0; j<deleteTemplate[i]->count; j++)   
        {
            // 为模版的坐标数据赋值。
            deleteTemplate[i]->tplData[j] = _defTemplateData[i][j*2];
            deleteTemplate[i]->tplData[j] = _defTemplateData[i][j*2+1];

            // 为模版的附属数据赋值。
            tplCudaHost[i]->attachedData[j] = _defTemplateAttachedData[i][j];   
        }    
    }

    // 将细化查询表内模版的坐标数据和附属数据拷贝到 device 端。
    for (int i = 0; i < lookTableLen; ++i)
    {
        errcode = TemplateBasicOp::copyToCurrentDevice(lookTable[i]);
        if (errcode != NO_ERROR)
            return errcode;
    }

    // 将模版数组的坐标数据和附属数据拷贝到 device 端。
    for(int i=0; i<deleteTemplateLen; i++)
    { 
        errcode = TemplateBasicOp::copyToCurrentDevice(deleteTemplate[i]);
        if (errcode != NO_ERROR)
            return errcode; 
    }

    // 因为 copyToCurrentDevice 函数只能将模版的坐标数据和附属数据拷贝到 device 端，
    // 而其他如 count 等数据还在 host 端，则需要一个中间变量将其他数据也拷到 device 端
    // 通过 lookTableCudaTemp 将细化查询表的其他数据拷到 device 端。
    for (int i = 0; i < lookTableLen; ++i)
    {
        hipMalloc((void **)&lookTableCudaTemp[i],
                    sizeof(TemplateCuda) );
        hipMemcpy( lookTableCudaTemp[i] , 
                    lookTableCudaHost[i],
                    sizeof(TemplateCuda),
                    hipMemcpyHostToDevice );
    }

    hipMalloc((void **)&tplCudaTemp, 
                sizeof(TemplateCuda) * deleteTemplateLen);
    // 通过 tplCudaTemp 将细化查询表的其他数据拷到 device 端。
    for(int i=0; i<deleteTemplateLen; i++)
    {
		
		hipMemcpy(tplCudaTemp[i], tplCudaHost[i], 
                            sizeof(TemplateCuda), 
                            hipMemcpyHostToDevice);
    }

    // 将模版的坐标数据、附属数据和其他数据拷进 device 端后，指向每个模版的指针数据还在 host 端，
    // 通过 lookTableCuda 将细化查询表里指向模版的指针数据拷进 device 端。
    hipMalloc((void **)&lookTableCuda,
                sizeof(TemplateCuda *) * lookTableLen );
    hipMemcpy (lookTableCuda, lookTableCudaTemp,
                sizeof(TemplateCuda *) * lookTableLen,
                hipMemcpyHostToDevice );
    
    // 通过 tplCuda 将删除模版数组里指向模版的指针数据拷进 device 端。
    hipMalloc((void **)&tplCuda, 
                sizeof(TemplateCuda *) * deleteTemplateLen);
    hipMemcpy( tplCuda, tplCudaTemp, 
                sizeof(TemplateCuda *) * deleteTemplateLen, 
                hipMemcpyHostToDevice);

    // 为标记数组开空间，大小与图像大小相同。
    hipMalloc((void **)&TabArray,
                inimg->width * inimg->height);
    
    // 为标记迭代的标记变量开空间。
    hipMalloc((void **)&dev_isIteration,
                sizeof(bool));

    // 提取输出图像
    ImageCuda outsubimgCud;
    errcode = ImageBasicOp::roiSubImage(outimg, &outsubimgCud);
    if (errcode != NO_ERROR)
        return errcode;

    // blocksize 使用默认的尺寸
    blocksize.x = DEF_BLOCK_X;
    blocksize.y = DEF_BLOCK_Y;

    // 使用最普通的方法划分 Grid 。
    gridsize.x = (outsubimgCud.imgMeta.width + blocksize.x - 1) / blocksize.x;
    gridsize.y = (outsubimgCud.imgMeta.height + blocksize.y - 1) / blocksize.y;
    
    // 为 isIteration 赋值为 true。
    *isIteration = true;

    // 开始迭代
    while(*isIteration)
    {
        // 赋值 isIteration 为 false。
        *isIteration = false;

        // 每次迭代都将 dev_isIteration 赋值为 false。
         hipMemcpy(dev_isIteration, isIteration,
                sizeof(bool),
                hipMemcpyHostToDevice);
                
                
        // 调用 Kernel 函数进行细化操作。
        _thinningKer<<<gridsize, blocksize>>>(outsubimgCud, 
                                      lookTableCuda,
                                      tplCuda, 
                                      deleteTemplateLen,
                                      lookTableLen,
                                      TabArray
                                      );
        if (hipGetLastError() != hipSuccess)
            return CUDA_ERROR;

        // 调用 Kernel 函数进行删除图像操作。
        _deleteImageKer<<<gridsize, blocksize>>>(outsubimgCud, TabArray, dev_isIteration);
        if (hipGetLastError() != hipSuccess)
            return CUDA_ERROR;
        
        // 将 dev_isIteration 的值拷到 host 端并赋值给 isIteration ，从而判断是否继续迭代。
        hipMemcpy(isIteration, dev_isIteration,
                sizeof(bool),
                hipMemcpyDeviceToHost);
    }

    // 迭代结束，释放前面申请的空间防止内存能泄露。
    hipFree(tplCuda);
    for (int i = 0; i < deleteTemplateLen; ++i)
        hipFree(tplCudaTemp[i]);

    hipFree(lookTableCuda);
    for (int i = 0; i < lookTableLen; ++i)
        hipFree(lookTableCudaTemp[i]);
        
    hipFree(TabArray);
    hipFree(dev_isIteration);

    // 将输出图像拷贝到 Host 上。
    errcode = ImageBasicOp::copyToHost(outimg);
    return errcode;
}

__host__ int Thinning::thin_MATLABLike(
            Image *inimg,    // 输入图像
            Image *outimg    // 输出图像
    )
{
    int errcode;  // 局部变量，错误码。
    dim3 gridsize;
    dim3 blocksize;
    int *changedCount = new int();
    int *dev_changedCount;
    
    
    
    // cout << *changedCount << endl;
    
    hipMalloc((void **)&dev_changedCount, sizeof(int));
    
    
    char *lutthin1_dev;
    char *lutthin2_dev;
    char *lutthin3_dev;
    char *lutthin4_dev;
//cout << "1" << endl;
    hipMalloc((void **)&lutthin1_dev, 512);
    hipMalloc((void **)&lutthin2_dev, 512);
    hipMalloc((void **)&lutthin3_dev, 512);
    hipMalloc((void **)&lutthin4_dev, 512);

    hipMemcpy (lutthin1_dev, lutthin1,
                512,
                hipMemcpyHostToDevice );
    hipMemcpy (lutthin2_dev, lutthin2,
                512,
                hipMemcpyHostToDevice );
    hipMemcpy (lutthin3_dev, lutthin3,
                512,
                hipMemcpyHostToDevice );
    hipMemcpy (lutthin4_dev, lutthin4,
                512,
                hipMemcpyHostToDevice );

//cout << "2" << endl;
    Image *tempimg;
    ImageBasicOp::newImage(&tempimg);
    ImageBasicOp::makeAtCurrentDevice(tempimg, inimg->width, inimg->height);

    ImageBasicOp::copyToCurrentDevice(inimg,outimg);

    // 提取输出图像
    ImageCuda outsubimgCud;
    errcode = ImageBasicOp::roiSubImage(outimg, &outsubimgCud);
    if (errcode != NO_ERROR)
        return errcode;

    // 提取输出图像
    ImageCuda tempsubimgCud;
    errcode = ImageBasicOp::roiSubImage(tempimg, &tempsubimgCud);
    if (errcode != NO_ERROR)
        return errcode;

    // blocksize 使用默认的尺寸
    blocksize.x = DEF_BLOCK_X;
    blocksize.y = DEF_BLOCK_Y;

    // 使用最普通的方法划分 Grid 。
    gridsize.x = (outsubimgCud.imgMeta.width + blocksize.x - 1) / blocksize.x;
    gridsize.y = (outsubimgCud.imgMeta.height + blocksize.y - 1) / blocksize.y;

    *changedCount = 1;
//cout << "3" << endl;
    while(*changedCount)
     //for(int i=0; i<5; i++)
    {
    *changedCount = 0;
    hipMemcpy (dev_changedCount, changedCount,
                sizeof(int),
                hipMemcpyHostToDevice );
        
//cout << "4" << endl;
        _thin_MATLABLike_FsubKer<<<gridsize, blocksize>>>(
        outsubimgCud,
        tempsubimgCud,
        lutthin1_dev,
        lutthin2_dev,
        lutthin3_dev
        );
// cout << "5" << endl;
         _thin_MATLABLike_SsubKer<<<gridsize, blocksize>>>(
        tempsubimgCud,
        outsubimgCud,
        lutthin1_dev,
        lutthin2_dev,
        lutthin4_dev,
        dev_changedCount
        );
        // cout << "6" << endl;
    
    hipMemcpy ( changedCount,dev_changedCount,
                sizeof(int),
                hipMemcpyDeviceToHost );
                // cout << *changedCount << "j"<<endl;
    
   }

    



    hipFree(lutthin1_dev);
    hipFree(lutthin2_dev);
    hipFree(lutthin3_dev);
    hipFree(lutthin4_dev);

    ImageBasicOp::deleteImage(tempimg);
	return 0;
  
}

           __host__ int 
    Thinning::_ser_thin_MATLABLike(
            // Image *inimg,    // 输入图像
            Image *inimg    // 输出图像
    ){
    char * workImg = new char[40000];

    unsigned int changedCount = 0;
    
    

    for(int s = 1; s < 199; s++) {// 境界上のpix に3-by-3 近隣がないに注意！  従って、その3-by-3 近隣パターンに対応する２進値もない。
                                       //注意附近没有 像素边界上的pix 3 - by - 3!因此,3 - by - 3附近方式应对2进值也达到了。
            for(int e = 1; e < 199; e++) {// imageの(1,1)に対応するindexは存在しない！  不要图像（1,1）对应的目录
                int S = s;// + s0;
                int E = e;// + e0;
                int index = 0;
                if(inimg->imgData[(S-1)*200 + E-1] > 0) index += 1;
                if(inimg->imgData[(S  )*200 + E-1] > 0)   index += 2;
                if(inimg->imgData[(S+1)*200 + E-1] > 0) index += 4;
                
                if(inimg->imgData[(S-1)*200 + E] > 0)   index += 8;
                if(inimg->imgData[(S  )*200 + E] > 0)     index += 16;
                if(inimg->imgData[(S+1)*200 + E] > 0)   index += 32;
                
                if(inimg->imgData[(S-1)*200 + E+1] > 0) index += 64;
                if(inimg->imgData[(S  )*200 + E+1] > 0)   index += 128;
                if(inimg->imgData[(S+1)*200 + E+1] > 0) index += 256;

                unsigned char replacedPix1 = lutthin1[index];
                unsigned char replacedPix2 = lutthin2[index];
                unsigned char replacedPix3 = lutthin3[index];

                workImg[s*200 + e] = inimg->imgData[S*200 + E] && !(replacedPix1 && replacedPix2 && replacedPix3);

            }

    }


    for(int s = 1; s < 199; s++) {// 境界上のpix に3-by-3 近隣がないに注意！  従って、その3-by-3 近隣パターンに対応する２進値もない。
        // 注意3-3接近的边界像素的情况下！因此，不存在对应的二进制值的3-3模式，接近。
            for(int e = 1; e < 199; e++) {// imageの(1,1)に対応するindexは存在しない！ 不存在索引，对应于（1,1）的图像！
                int index = 0;
                if(workImg[(s-1)*200 + e-1] > 0) index += 1;
                if(workImg[(s  )*200 + e-1] > 0)   index += 2;
                if(workImg[(s+1)*200 + e-1] > 0) index += 4;
                
                if(workImg[(s-1)*200 + e] > 0)   index += 8;
                if(workImg[(s  )*200 + e] > 0)     index += 16;
                if(workImg[(s+1)*200 + e] > 0)   index += 32;
                
                if(workImg[(s-1)*200 + e+1] > 0) index += 64;
                if(workImg[(s  )*200 + e+1] > 0)   index += 128;
                if(workImg[(s+1)*200 + e+1] > 0) index += 256;

                unsigned char replacedPix1 = lutthin1[index];
                unsigned char replacedPix2 = lutthin2[index];
                unsigned char replacedPix4 = lutthin4[index];

                unsigned char niv = workImg[s*200 + e] && !(replacedPix1 && replacedPix2 && replacedPix4);

                int S = s;
                int E = e;// + e0;

                if(niv != (inimg->imgData[(S)*200 + E] > 0)) {
		    if(niv > 0)
			inimg->imgData[(S)*200 + E] = 255;//(niv>0)?255:0;
                    else
			inimg->imgData[(S)*200 + E] = niv;
                    changedCount++;
                }


            }

        }
        // cout << changedCount << "f" << endl;

    return  changedCount;
    
    }
               __host__ int 
    Thinning::ser_thin_MATLABLike(
            Image *inimg,    // 输入图像
            Image *outimg    // 输出图像
    )
    {
    // cout << "1" << endl;
	ImageBasicOp::copyToHost(inimg,outimg);
	
	// cout << "1" << endl;
		while(_ser_thin_MATLABLike(outimg) > 0);
	return 0;
    }
    


