#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

/*#include "Image.h"
#include "Template.h"
#include "AffineTrans.h"
#include "RotateTable.h"
#include "SelectShape.h"
#include "SortArray.h"
#include "ErrorCode.h"
#include "Histogram.h"
#include "HistogramSpec.h"
#include "Threshold.h"
#include "Binarize.h"
#include "Morphology.h"
#include "ConnectRegion.h"
#include "Flip.h"
#include "BoundingRect.h"
#include "FillUp.h"
#include "Moments.h"
#include "GeometryProperties.h"
#include "ImageShrink.h"
#include "HoughLine.h"
#include "CoordiSet.h"
#include "FeatureVecCalc.h"
#include "FeatureVecArray.h"
#include "ScanArray.h"*/
#include "Thinning.h"

#define ROUND_NUM 1

int main()
{
//-------WorkAndObjectMatch------------//
    // Image **images = new Image*[2];
    // int imagecount = 2;
    // ImageBasicOp::newImage(&images[0]);
    // ImageBasicOp::readFromFile("A8_64.bmp", images[0]);
    // ImageBasicOp::newImage(&images[1]);
    // ImageBasicOp::readFromFile("A8_128.bmp", images[1]);

    // Image *test;
    // ImageBasicOp::newImage(&test);
    // ImageBasicOp::readFromFile("test8_abc.bmp", test);

    // RotateTable rotatetable;
    // rotatetable.setMinAngle(10);
    // rotatetable.setMaxAngle(20);
    // rotatetable.setDetAngle(1);
    // rotatetable.setSizeX(images[0]->width * 2);
    // rotatetable.setSizeY(images[0]->height * 2);
    // cout << "min angle " << rotatetable.getMinAngle() << endl;
    // cout << "max angle " << rotatetable.getMaxAngle() << endl;

   // int dwidth = 32;
   // int dheight = 32;
   // int dx = 32;
   // int dy = 32;
    // int dwidth = images[0]->width * 1.5;
    // int dheight = images[0]->height * 1.5;
    // int dx = dwidth / 2;
    // int dy = dheight / 2;

    // RotateTable worktable;
    // worktable.setMinAngle(10);
    // worktable.setMaxAngle(20);
    // worktable.setDetAngle(1);
    // worktable.setSizeX(images[0]->width / 8 * 2);
    // worktable.setSizeY(images[0]->height / 8 * 2);

    // ImagesInfo workinfo;
    // workinfo.images = images;
    // workinfo.count = imagecount;
    // workinfo.rotateTable = &worktable;
    // workinfo.dWidth = test->width / 8;
    // workinfo.dHeight = test->height / 8;
    // workinfo.dX = workinfo.dWidth / 2;
    // workinfo.dY = workinfo.dHeight / 2;

    // ImagesInfo imagesinfo[2];
    // int imagesinfocount = 1;
    // for (int i = 0; i < imagesinfocount; i++) {
        // imagesinfo[i].images = images;
        // imagesinfo[i].count = imagecount;
        // imagesinfo[i].rotateTable = &rotatetable;
        // imagesinfo[i].dWidth = dwidth;
        // imagesinfo[i].dHeight = dheight;
        // imagesinfo[i].dX = dx;
        // imagesinfo[i].dY = dy;
    // }
   // Image *test;
   // ImageBasicOp::newImage(&test);
   // ImageBasicOp::readFromFile("test8_ab.bmp", test);
    // WorkAndObjectMatch objectmatch(&workinfo, imagesinfo, imagesinfocount,
                                   // 10);
    // MatchRes res[2];
    // int rescount = 1;
    // int errcode;
    // float rumtime;
    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // hipEventRecord(start, 0);
    // errcode = objectmatch.workAndObjectMatch(test, res, rescount);
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&rumtime, start, stop);
    // if (errcode != NO_ERROR) {
        // cout << "error code is " << errcode << endl;
        // return -1;
    // }
    // cout << "success" << endl;
    // cout << "the time is " << (rumtime / ROUND_NUM) << " ms" << endl;
    // cout << "res:" << endl;
    // for (int i = 0; i < rescount; i++) {
        // cout << i << ":" << res[i].tplIndex << endl;
    // }

    // return 0;
    // RotateTable rt;
    // rt.setMinAngle(15);
    // rt.setMaxAngle(15);
    // rt.setDetAngle(0.2);
    // cout << rt.getAngleCount() << endl;
    // for (int i = 0; i < rt.getAngleCount(); i++)
        // cout << rt.getAngleVal(i) << ", ";
    // cout << endl;
    // cout << rt.getAngleIdx(15.05) << " " << rt.getAngleIdx(15.37) << endl;

        // Image *img, *img2;
//----Affine Transform----//    
    // AffineTrans at;
    // at.setAlpha(60.0f);
    // at.setX(320);
    // at.setY(240);
    // at.setImpType(SOFT_IPL);
    // //cout << at.getAlpha() << endl;
    
//----Select Shape----//
//    int areaRank[14] = {72266,127,73248,82,37966,195,61483,242,73248,215,74760,192,68600,42};
//    int pairsNum = 7;
//    int index = 2;
//    int value = 73248;
//    int minValue = 73248;
//    int maxValue = 74760;
//    SelectShape ss;
//    ss.setRank(areaRank);
//    ss.setPairsNum(pairsNum);
//    ss.setIndex(index);
//    ss.setValue(value);
//    ss.setMinValue(minValue);
//    ss.setMaxValue(maxValue);
    
    
//----Sort Array----//
//    SortArray sa;
//    int inarray[8] = {4,5,7,2,1,3,8,6};
//    int outarray[8] = {0};
//    sa.setLength(8);
//    sa.setLenSec(1);
//    sa.setSortFlag(1);
//    sa.setIsHost(1);
    
//    sa.bitonicSort(inarray, outarray);  
//    for (int i = 0; i < 8; i++)
//    {
//         cout<<outarray[i]<<endl;
//    }
  
//----Histogram----
//    Image *inimg;
//    ImageBasicOp::newImage(&inimg);
    
//    cout << "AA" << endl;
    
//    ImageBasicOp::readFromFile("hist_in.bmp", inimg);
    
//    Histogram hist;
//    unsigned int histogram[256] = {0};
//    bool onhostarray = true;
//    hist.histogram(inimg, histogram, onhostarray);
//    unsigned int sum = 0;
    
//    for (int i = 0; i < 256; i++)
//    {
//        sum += histogram[i];
//        cout << i << "=" << histogram[i] << endl;
//    }
    
//    cout << "The sum pixel is: " << sum << endl;
    
//----HistogramSpec----
//    Image *inimg;
//    ImageBasicOp::newImage(&inimg);
//    ImageBasicOp::readFromFile("hist_in.bmp", inimg);
    
//    Image *outimg;
//    ImageBasicOp::newImage(&outimg);
//    ImageBasicOp::makeAtHost(outimg, inimg->width, inimg->height);
    
//    Image *refimg;
//    ImageBasicOp::newImage(&refimg);
//    ImageBasicOp::readFromFile("hist_ref.bmp", refimg);
       
//    float histspec[256];
    
//    for (int j = 0; j < 256; j++) {
//        histspec[j] = (float)1 / 256;
//    }
    
//    HistogramSpec hs;
//    hs.setRefimg(refimg);
//    hs.setRefHisto(histspec);
//    //hs.HistogramEquilibrium(inimg, outimg);
//    //hs.HistogramSpecByImage(inimg, outimg);
//    hs.HistogramSpecByHisto(inimg, outimg);
    
//    ImageBasicOp::copyToHost(outimg);
//    //ImageBasicOp::writeToFile("hist_outequ.bmp", outimg);
//    //ImageBasicOp::writeToFile("hist_outimg.bmp", outimg);
//    ImageBasicOp::writeToFile("hist_outhist.bmp", outimg);
    
//    ImageBasicOp::deleteImage(inimg);
//    ImageBasicOp::deleteImage(outimg);
//    ImageBasicOp::deleteImage(refimg);
   
    
//    ImageBasicOp::newImage(&img);
//    ImageBasicOp::newImage(&img2);
    
//    cout << "AA" << endl;

//    ImageBasicOp::readFromFile("selectshape_in.bmp", img);
//    ImageBasicOp::readFromFile("selectshape_in.bmp", img2);
    
//    //img->roiX1 = 240;
//    //img->roiY1 = 50;

//    //cout<<ss.selectShapeByIndex(img, img2)<<endl;
//    //ImageBasicOp::makeAtCurrentDevice(img2, 640, 480);

//    //img2->roiY1 = 100;

//    //cout << "xx" << endl;

//    // cout << at.rotateCenter(img, img2) << endl;
//    //ImageBasicOp::copyToCurrentDevice(img, img2);
//    cout <<hipGetErrorString(hipGetLastError())<<endl;

//    //ImageBasicOp::writeToFile("selectshape_out.bmp", img2);

//    ImageBasicOp::deleteImage(img);
//    ImageBasicOp::deleteImage(img2);

//----LabelIslandSortArea----
//      Image *inimg;
//      ImageBasicOp::newImage(&inimg);
//      ImageBasicOp::readFromFile("selectshape_in.bmp", inimg);
//      unsigned int areaRank[256*2];
//      LabelIslandSortArea lisa;
//      lisa.setIsHost(1);
//      lisa.setMinArea(100);
//      lisa.setMaxArea(10000000);
//      lisa.setSortFlag(1);
//      lisa.labelIslandSortArea(inimg, areaRank);
      
//      for (int i=0;i<lisa.getLength();i++)
//      {
//           cout<<areaRank[2*i]<<"\t"<<areaRank[2*i+1]<<endl;
//      }

//----Binarize----
    // Image *inimg;
    // ImageBasicOp::newImage(&inimg);
    // ImageBasicOp::readFromFile("okano01.bmp", inimg);
    
    // Image *outimg;
    // ImageBasicOp::newImage(&outimg);
    // ImageBasicOp::makeAtHost(outimg, inimg->width, inimg->height);
    
	// unsigned char threshold = 20;
	// Binarize bi;
	// bi.setThreshold(threshold);
	
	// cout << "AA" << endl;
	// bi.binarize(inimg, outimg);
    
    // ImageBasicOp::copyToHost(outimg);
    // ImageBasicOp::writeToFile("okana02.bmp", outimg);
    
    // ImageBasicOp::deleteImage(inimg);
    // ImageBasicOp::deleteImage(outimg);

    // cout << "BB" << endl;
    
    // return 0;
	
	// --- ConnectRegion ---
//    Image *inimg;
//    ImageBasicOp::newImage(&inimg);
//    ImageBasicOp::readFromFile("okano01.bmp", inimg);
    
//    Image *outimg;
//    ImageBasicOp::newImage(&outimg);
//    ImageBasicOp::makeAtHost(outimg, inimg->width, inimg->height);
    
//    int threshold = 1;
//    int minArea = 1300;
//    int maxArea = 60000;
//    ConnectRegion cr;
//    cr.setThreshold(threshold);
//    cr.setMinArea(minArea);
//    cr.setMaxArea(maxArea);
	
//    cout << "AA" << endl;
//    cr.connectRegion(inimg, outimg);
    
//    ImageBasicOp::copyToHost(outimg);
//    ImageBasicOp::writeToFile("okano02.bmp", outimg);
    
//    ImageBasicOp::deleteImage(inimg);
//    ImageBasicOp::deleteImage(outimg);

//    cout << "BB" << endl;

// ----Flip----
//    int i;
//    hipEvent_t start, stop;
//    hipEventCreate(&start);
//    hipEventCreate(&stop);
//    float runTime;
//
//    Image *inimg;
//    ImageBasicOp::newImage(&inimg);
//    ImageBasicOp::readFromFile("okano01.bmp", inimg);
//
//    Image *outimg;
//    ImageBasicOp::newImage(&outimg);
//    ImageBasicOp::makeAtHost(outimg, inimg->width, inimg->height);
//
//    Flip fi;
//    
//    cout << "123" << endl;
//
//    hipEventRecord(start, 0);
//    for (i = 0; i < ROUND_NUM; i++) {
//        fi.flipHorizontal(inimg,outimg);
//    }
//    hipEventRecord(stop, 0);
//    hipEventSynchronize(stop);
//    hipEventElapsedTime(&runTime, start, stop);
//
//    cout << "The total H_time is " << (runTime/ROUND_NUM) << " ms" << endl;
//
//    hipEventRecord(start, 0);
//    for (i = 0; i < ROUND_NUM; i++) {
//        fi.flipVertical(inimg,outimg);
//    }
//    hipEventRecord(stop, 0);
//    hipEventSynchronize(stop);
//    hipEventElapsedTime(&runTime, start, stop);
//
//    cout << "The total V_time is " << (runTime/ROUND_NUM) << " ms" << endl;
//
//    ImageBasicOp::copyToHost(outimg);
//    ImageBasicOp::writeToFile("okano02.bmp", outimg);
//
//    ImageBasicOp::deleteImage(inimg);
//    ImageBasicOp::deleteImage(outimg);
//
//    cout << "BB" << endl;
//    hipEventDestroy(start);
//    hipEventDestroy(stop);
//    return 0;
//}
// ----BoundingRect----
//   Image *inimg;
//    ImageBasicOp::newImage(&inimg);
//    ImageBasicOp::readFromFile("00.bmp", inimg);
    
//    unsigned char value = 253;
//    BoundingRect br;
//    Quadrangle qr;
//    DirectedRect dr;
//    br.setValue(value);
//    cout << "AA" << endl;
//    br.boundingRect(inimg, &qr);
	   
 //   cout << "Quadrangle information:" << endl;
//    cout << "angle:"<< qr.angle << endl;
 //   cout << "The four vertex:" << endl;
//    cout << "(" << qr.points[0][0] << "," << qr.points[0][1] << ")" << endl;
//    cout << "(" << qr.points[1][0] << "," << qr.points[1][1] << ")" << endl; 
 //   cout << "(" << qr.points[2][0] << "," << qr.points[2][1] << ")" << endl;
//    cout << "(" << qr.points[3][0] << "," << qr.points[3][1] << ")" << endl;
		
 //   br.boundingRect(inimg, &dr);
//    cout << "DirectedRect information:" << endl;
//    cout << "angle:" << qr.angle << endl;
//    cout << "center point is:" << dr.centerPoint[0] << "," << dr.centerPoint[1] 
 //        << endl;
 //   cout << "length1 is:" << dr.length1 << endl;
//    cout << "length2 is:" << dr.length2 << endl;
    
//    ImageBasicOp::deleteImage(inimg);
//
//    cout << "BB" << endl;

	//----FillUp----
/*
   Image *inimg;
    ImageBasicOp::newImage(&inimg);
    ImageBasicOp::readFromFile("fillup_in.bmp", inimg);
   
    Image *outimg;
    ImageBasicOp::newImage(&outimg);
    ImageBasicOp::makeAtHost(outimg, inimg->width, inimg->height);
    
    unsigned char l = 255;
    unsigned char v = 0;
   int maxw = 15;
   float r = 0.2;
  int stateflag = 0;
    Template *tm;
     
    TemplateBasicOp::newTemplate(&tm);
    TemplateBasicOp::makeAtHost(tm, maxw *maxw);
   
    for (int i = 0; i < maxw *maxw; i++) {
       
       tm->tplData[2 * i] = i % maxw-maxw / 2;
       tm->tplData[2 * i + 1] = i / maxw-maxw / 2;
   }

    FillUp fl;
    fl.setL(l);
    fl.setV(v);
    fl.setMaxw(maxw);
    fl.setR(r);
    fl.setTemplate(tm);
	
    cout << "AA" << endl;
   fl.fillUp(inimg, outimg);
  
    //fl.fillUpAdv(inimg, outimg, &stateflag);
    
    //cout << "stateflag = " << stateflag << endl;
   cout << "BB" << endl;
    
    ImageBasicOp::writeToFile("fillup_out.bmp", outimg);
    
    ImageBasicOp::deleteImage(inimg);
   ImageBasicOp::deleteImage(outimg);

     cout << "CC" << endl;
    
    // return 0;
*/
// ----Moments----
/*
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float runTime;
    int round = 1;
    
    Image *inimg;
    ImageBasicOp::newImage(&inimg);
    ImageBasicOp::readFromFile("momentsImage.bmp", inimg);

    Moments mom;
    MomentSet mset;
    mom.setIsconst(false);
    //float centers[2];
    //float angle;
    hipEventRecord(start, 0);
    int i = 0;
    while(i < round ) {
         cout<<mom.spatialMoments(inimg, &mset)<<endl;
         i++;
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&runTime, start, stop);
    cout << "The total time is " << (runTime/round) << " ms" << endl;

    //cout<<"the centers is "<<centers[0]<<"\t"<<centers[1]<<endl;
    //cout<<"the angle is "<<angle<<endl;
    cout<<mset.m00<<endl;
    cout<<mset.m10<<endl;
    cout<<mset.m01<<endl;
    cout<<mset.m20<<endl;
    cout<<mset.m11<<endl;
    cout<<mset.m02<<endl;
    cout<<mset.m30<<endl;
    cout<<mset.m21<<endl;
    cout<<mset.m12<<endl;
    cout<<mset.m03<<endl;
*/

//// ---- GeometryProperties ----   

//    hipEvent_t start, stop;
//    hipEventCreate(&start);
//    hipEventCreate(&stop);
//    float runTime;
//    int round = 100;
    
//    Image *inimg;
//    ImageBasicOp::newImage(&inimg);
//    ImageBasicOp::readFromFile("momentsImage.bmp", inimg);

//    GeometryProperties geoprop;
//    float length = 0.0f;
    
//    int i = 0;
//    while(i < round ) {
//    hipEventRecord(start, 0);
//         cout<<geoprop.contourLength(inimg, &length)<<endl;
//         i++;
    
//    hipEventRecord(stop, 0);
//    hipEventSynchronize(stop);
//    hipEventElapsedTime(&runTime, start, stop);
//    cout << "The  time is " << runTime<< " ms" << endl;
//    }
//    cout<<"the contour length is: "<<length<<endl;
    
//    ImageBasicOp::deleteImage(inimg);
//    hipEventDestroy(start);
//    hipEventDestroy(stop);

//    //----ImageShrink----
//    int times = 2;
//    ImageShrink shi;
//    shi.setTimes(times);
    
//    hipEvent_t start, stop;
//    float elapsedTime = 0.0;
    
//    Image *inimg;
//    ImageBasicOp::newImage(&inimg);
//    ImageBasicOp::readFromFile("okano01.bmp", inimg);
    
//    Image *outimg;
//    ImageBasicOp::newImage(&outimg);
//    ImageBasicOp::makeAtHost(outimg, inimg->width / times, 
//                             inimg->height / times);
	
//    cout << "AA" << endl;
    
//    hipEventCreate(&start);
//    hipEventCreate(&stop);
//    cout << "Test start!" << endl;
    
//    hipEventRecord(start, 0);
    
//    //shi.imageshrinkbyDom(inimg, outimg);
//    shi.imageshrinkbyPro(inimg, outimg);
    
//    hipEventRecord(stop, 0);
//    hipEventSynchronize(stop);
//    hipEventElapsedTime(&elapsedTime, start, stop);

//    cout << "Test end!" << endl;
//    cout << "Execute time:" << (float)elapsedTime << "ms" << endl;
    
    
//    hipEventDestroy(start);
//    hipEventDestroy(stop);
    
//    ImageBasicOp::copyToHost(outimg);
//    ImageBasicOp::writeToFile("imageshrink_out.bmp", outimg);
    
//    ImageBasicOp::deleteImage(inimg);
//    ImageBasicOp::deleteImage(outimg);

    //-------HoughLine-------//
//    Image *inimg;
//    ImageBasicOp::newImage(&inimg);
//    ImageBasicOp::readFromFile("hough_in4.bmp", inimg); 
    
//    Image *outimg;
//    ImageBasicOp::newImage(&outimg);
//    ImageBasicOp::makeAtHost(outimg, inimg->width, inimg->height);
    
//    cout << "aa" << endl;
//    HoughLine hough;
    
//    double detheta = 1;
//    int threshold = 200;
//    int linenum = 10;
//    int thetasize = 5;
//    int rhosize = 10;
    
    
//    CoordiSet *coor;
//    coor = NULL;
    /*
    CoordiSetBasicOp::newCoordiSet(&coor);
    
    CoordiSetBasicOp::makeAtHost(coor, 1098);    
    
    int k = 0;
    for (int j = 0; j < inimg->height; j++) {
        for (int i = 0; i < inimg->width; i++) {
         
            int index = j * inimg->width + i;
            if (inimg->imgData[index] == 255) {
                coor->tplData[2 * k] =  i;
                coor->tplData[2 * k + 1] = j;  
                k++;
            }
        } 
    }
    */
    /*
    hough.setDeTheta(detheta);
    hough.setThreshold(threshold);
    hough.setLineNum(linenum);
    hough.setThetaSize(thetasize);
    hough.setRhoSize(rhosize);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float runTime;
   
    hipEventRecord(start, 0); 
    
    hough.houghline(inimg, coor, outimg);  

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&runTime, start, stop);

    cout << "The total time is " << runTime << " ms" << endl;

    ImageBasicOp::copyToHost(outimg);
    ImageBasicOp::writeToFile("hough_out.bmp", outimg);
    
    ImageBasicOp::deleteImage(inimg);
    ImageBasicOp::deleteImage(outimg);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);

    cout << "CC" << endl;    
    
    return 0;
    */
     // ---- FeatureVector ----

    // cout << "BB" << endl;

    // Image *inimg;
    // ImageBasicOp::newImage(&inimg);
    // ImageBasicOp::readFromFile("okano01.bmp", inimg);
    // int imgwidth = inimg->width;
    // int imgheight = inimg->height;
    
    ////选取一个 5 * 5 大小的方形区域作为坐标集
    // int width = 5; 
    // CoordiSet *incoordiset;
    // CoordiSetBasicOp::newCoordiSet(&incoordiset);
    // CoordiSetBasicOp::makeAtHost(incoordiset, width * width);

    // int x = imgwidth / 2 - width / 2;
    // int y = imgheight / 2 - width / 2;

    ////初始化坐标集，选取坐标中心
    // for (int i = 0; i < width * width; i++) {
       
       // incoordiset->tplData[2 * i] = x;
       // incoordiset->tplData[2 * i + 1] = y;

       // x++;
       // if ((i + 1) % width == 0){
           // x = imgwidth / 2 - width / 2;
           // y++;
       // }
   // }

   // FeatureVecArray  outfeaturevecarray;
   // FeatureVecArrayBasicOp::makeAtCurrentDevice(&outfeaturevecarray, 
                                               // width * width);
   
   // cout << outfeaturevecarray.count << endl;
   // if (&outfeaturevecarray == NULL)
       // cout << "NULL" << endl;
   // FeatureVecCalc featureveccalc (0.1, 0.1, 4, 2);
   // hipEvent_t start, stop;
   // hipEventCreate(&start);
   // hipEventCreate(&stop);
   // float runTime;
   // hipEventRecord(start, 0);
   // featureveccalc.calFeatureVector(inimg, incoordiset, &outfeaturevecarray);
   // hipEventRecord(stop, 0);
   // hipEventSynchronize(stop);
   // hipEventElapsedTime(&runTime, start, stop);
   // cout << "The  time is " << runTime<< " ms" << endl;
    
   
    
   // ImageBasicOp::deleteImage(inimg);
   
   // TemplateBasicOp::deleteTemplate(incoordiset);
   // FeatureVecArrayBasicOp::deleteFeatureVecArray(&outfeaturevecarray);
   
   // cout << "CC" << endl;

   // return 0;

// -----SalientRegionDetect-----
/*
    hipEvent_t start, stop;
    float elapsedTime = 0.0;

    Image *inimg, *outimg;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    ImageBasicOp::newImage(&inimg);
    ImageBasicOp::readFromFile("okano01.bmp", inimg);
    //ImageBasicOp::readFromFile("hist_in.bmp", inimg);
    ImageBasicOp::newImage(&outimg);
    ImageBasicOp::makeAtHost(outimg, inimg->width, inimg->height);

    SalientRegionDetect srd;
    int radius[10] = {5, 8, 13};
    int smoothWidth[10] = {3, 5};
    srd.setHighPercent(0.1);
    srd.setLowPercent(0.3);
    srd.setIterationSM1(1);
    srd.setIsSelect(false);
    srd.setIterationSM2(1);
    srd.setRadius(radius);
    srd.setSmoothWidth(smoothWidth);
    srd.setMeanTpl(3);
    srd.setWeightSM1(1);
    srd.setWeightSM2(0);
    srd.setMinRegion(50);
    srd.setMaxRegion(10000);
    srd.setSaliencyThred(1);
    cout<<"Test start!"<<endl;
    int i = 0;
    while (i < 1) {
        hipEventRecord(start, 0);

        cout<<srd.saliencyRegionDetect(inimg, outimg)<<endl;
        
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        cout<<elapsedTime<<endl;

        i++;
    }
    cout<<"Test end!"<<endl;

    ImageBasicOp::writeToFile("saliencyMap_smooth.bmp", outimg);
    ImageBasicOp::deleteImage(inimg);
    ImageBasicOp::deleteImage(outimg);
    hipEventDestroy(start);
    hipEventDestroy(stop);
*/
//---------------RobustEdgeDetection---------------//
//    int i;
//    int k = 3;
//    hipEvent_t start, stop;
//    hipEventCreate(&start);
//    hipEventCreate(&stop);
//    float runTime;

//    Image *inimg;
//    ImageBasicOp::newImage(&inimg);
//    //ImageBasicOp::readFromFile("selectshape_in.bmp", inimg);
//    ImageBasicOp::readFromFile("hist_in.bmp", inimg);
//    //ImageBasicOp::readFromFile("1.bmp", inimg);

//    Image *outimg;
//    ImageBasicOp::newImage(&outimg);
//    ImageBasicOp::makeAtHost(outimg, inimg->width, inimg->height);

//    RobustEdgeDetection R;
    
//    R.setSearchScope(3);

//    cout << "123" << endl;

//    hipEventRecord(start, 0);
//    //int errcode;
//    for (i = 0; i < ROUND_NUM; i++) {
//        R.detectEdgeSA(k,inimg,outimg,NULL);
//    }
//    hipEventRecord(stop, 0);
//    hipEventSynchronize(stop);
//    hipEventElapsedTime(&runTime, start, stop);

//    cout << "The total H_time is " << (runTime/ROUND_NUM) << " ms" << endl;

////    hipEventRecord(start, 0);
////    for (i = 0; i < ROUND_NUM; i++) {
////        f1.detectEdgeFV(inimg,outimg);
////    }
////    hipEventRecord(stop, 0);
////    hipEventSynchronize(stop);
////    hipEventElapsedTime(&runTime, start, stop);
////
////    cout << "The total V_time is " << (runTime/ROUND_NUM) << " ms" << endl;
////
//    ImageBasicOp::copyToHost(outimg);
//    //ImageBasicOp::writeToFile("selectshape_out.bmp", outimg);
//    ImageBasicOp::writeToFile("hist_ref.bmp", outimg);
//    //ImageBasicOp::writeToFile("2.bmp", outimg);

//    ImageBasicOp::deleteImage(inimg);
//    ImageBasicOp::deleteImage(outimg);

//    cout << "BB" << endl;
//    hipEventDestroy(start);
//    hipEventDestroy(stop);
//    cout << "CC" << endl;

//-----------  OtsuBinarize  -------------
//     hipEvent_t start, stop;
//     hipEventCreate(&start);
//     hipEventCreate(&stop);
//     float runTime;
//     Image *inimg;
//     ImageBasicOp::newImage(&inimg);
//     ImageBasicOp::readFromFile("hist_in.bmp", inimg);
    
//     Image *outimg;
//     ImageBasicOp::newImage(&outimg);
//     ImageBasicOp::makeAtHost(outimg, inimg->width, inimg->height);
    
//     OtsuBinarize ob;
//     cout << "AA" << endl;
//     hipEventRecord(start, 0);
         
//     ob.otsuBinarize(inimg, outimg);
    
//     hipEventRecord(stop, 0);
//     hipEventSynchronize(stop);
//     hipEventElapsedTime(&runTime, start, stop);

     
//     cout << "The total H_time is " << (runTime/ROUND_NUM) << " ms" << endl;

//     ImageBasicOp::copyToHost(outimg);
//     ImageBasicOp::writeToFile("OtsuBinarize_out.bmp", outimg);
     
    
//     ImageBasicOp::deleteImage(inimg);
     
//     ImageBasicOp::deleteImage(outimg);
/*
//-------Scan-------
    unsigned int num_elements = 4096;
    const unsigned int mem_size = sizeof(float) * num_elements;

    // allocate host memory to store the input data
    float *inarray = (float*)malloc(mem_size);
    float *outarray = (float*)malloc(mem_size);
  
    for( unsigned int i = 0; i < num_elements; ++i) 
    {
        inarray[i] = 3;
        outarray[i] = 4;
    }

    ScanArray sa;
    sa.setScanType(BETTER_SCAN);

    bool inhost, outhost;
    inhost = true;
    outhost = true;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float runTime;
   
    hipEventRecord(start, 0);
    int i = 0;
    while(i < ROUND_NUM ) {
        sa.scanArray(inarray, outarray, num_elements, inhost, outhost);
        i++;
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&runTime, start, stop);
   
    for( unsigned int i = 0; i < num_elements; ++i) 
    {
        cout << "the outarray " << i << " is " << outarray[i] << endl;
    }
    
    cout << "The  time is " << (runTime/ROUND_NUM) << " ms" << endl;
   
    cout << "CC" << endl;

    return 0;
*/
//-------ScanLargeArray-------
//    unsigned int num_elements = 2048;
//    const unsigned int mem_size = sizeof(float) * num_elements;

//    // allocate host memory to store the input data
//    float *inarray = (float*)malloc(mem_size);
//    float *outarray = (float*)malloc(mem_size);
  
//    for( unsigned int i = 0; i < num_elements; ++i) 
//    {
//        inarray[i] = 2;
//        outarray[i] = 0;
//    }

//    ScanLargeArray sla;
//    bool inhost, outhost;
//    inhost = true;
//    outhost = true;
//    hipEvent_t start, stop;
//    hipEventCreate(&start);
//    hipEventCreate(&stop);
//    float runTime;
     
//    cout << "1111" << endl;
//    hipEventRecord(start, 0);
//    int i = 0;
//    while(i < ROUND_NUM ) {
//        sla.scanLargeArray(inarray, outarray, num_elements, inhost, outhost);
//        i++;
//    }
//    hipEventRecord(stop, 0);
//    hipEventSynchronize(stop);
//    hipEventElapsedTime(&runTime, start, stop);
   
//    for( unsigned int i = 0; i < num_elements; ++i) 
//    {
//        cout << "the outarray " << i << " is " << outarray[i] << endl;
//    }
    
//    cout << "The  time is " << (runTime/ROUND_NUM) << " ms" << endl;
//    cout << "CC" << endl;
//    return 0;

//------------------Thinning-----------------------//
    Thinning thin;
    
    CoordiSet *incst;
    CoordiSet *outcst;
    int num = 1000;
    int errcode;
    CoordiSetBasicOp::newCoordiSet(&incst);
    CoordiSetBasicOp::newCoordiSet(&outcst);
    errcode =CoordiSetBasicOp::makeAtHost(incst, num);
    errcode =CoordiSetBasicOp::makeAtHost(outcst, num);
    for (int i = 0; i < num; i += 1)
    {
        incst->tplData[2 * i] = 10;
        incst->tplData[2 * i + 1] = (i + 1) % 1000;
       
        
       

    }
    
    
    
    if (errcode != NO_ERROR) 
    {
        cout << "error: " << errcode << endl;
        return 0; 
    }

//    hipEvent_t start, stop;
//    hipEventCreate(&start);
//    hipEventCreate(&stop);
//    float runTime;
//    for (int i = 0; i < 100; i++) {
//        hipEventRecord(start, 0);
    
       cout<<thin.thinMatlabLike(incst, outcst)<<endl;
     
//        hipEventRecord(stop, 0);
//        hipEventSynchronize(stop);
//        hipEventElapsedTime(&runTime, start, stop);
//        cout << "The total H_time is " << (runTime) << " ms" << endl;
//    }
    
    CoordiSetBasicOp::deleteCoordiSet(incst);
    CoordiSetBasicOp::deleteCoordiSet(outcst);


    /*Image *inimg;
//    ImageBasicOp::newImage(&inimg);
    int errcode;
    errcode = ImageBasicOp::readFromFile("thinImg.bmp", inimg);
    if (errcode != NO_ERROR) 
    {
        cout << "error: " << errcode << endl;
        return 0; 
    }
    
    Image *outimg;
    ImageBasicOp::newImage(&outimg);
    ImageBasicOp::makeAtHost(outimg, inimg->width, inimg->height);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float runTime;
    for (int i = 0; i < 1; i++) {
		hipEventRecord(start, 0);
	
		cout<<thin.thinMatlabLike(inimg, outimg)<<endl;
	
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&runTime, start, stop);
		cout << "The total H_time is " << (runTime) << " ms" << endl;
    }

    ImageBasicOp::copyToHost(outimg);
    
    ImageBasicOp::writeToFile("thinningOut.bmp", outimg);  
    ImageBasicOp::deleteImage(inimg);
    ImageBasicOp::deleteImage(outimg);*/
    
    return 0;
}

